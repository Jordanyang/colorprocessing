#include "hip/hip_runtime.h"
#include <opencv2\opencv.hpp>
#include <iostream>
#define  WINDOW_SIZE 3
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <>
#include"..\\lib\\ImageNoise.h"
#include"..\\lib\\MyTime.h"
#include"..\\lib\\mycuda.h"
__device__ void setValue(uchar3* dst, int width, int x, int y, uchar3 value)
{
	int offset = y*width + x;
	dst[offset] = value;
}
__device__ uchar3 getValue(const uchar3* src, int width, int x, int y)
{
	int offset = y*width + x;
	return src[offset];
}
__device__ uchar sort_uchar(uchar* filter, int num)
{
	////Sorting in filter
	for (int i = 0; i <num; i++) {
		for (int j = i + 1; j < num; j++) {
			if (filter[i] > filter[j]) {
				//Swap the variables.
				uchar tmp = filter[i];
				filter[i] = filter[j];
				filter[j] = tmp;
			}
		}
	}
	uchar temp = filter[num / 2];
	return temp;
}
__device__ uchar3 sort(uchar3 *filter, int num)
{
	uchar3 temp;
	uchar a[9], b[9], c[9];
	for (int i = 0; i < num; i++)
	{
		a[i] = filter[i].x;
		b[i] = filter[i].y;
		c[i] = filter[i].z;
	}
	temp.x = sort_uchar(a, 9);
	temp.y = sort_uchar(b, 9);
	temp.z = sort_uchar(c, 9);
	return temp;

}
__global__ void Median_Filter_kernel(const uchar* src1, uchar* dst1, int width, int height)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	const uchar3 *src = (const uchar3 *)src1;
	uchar3 *dst = (uchar3*)dst1;
	//
	//filter mask
	uchar3 filter[9];

	if ((y >= height) || (x >= width))
		return;
	/////checking boundry conditions
	if ((y == 0) || (x == 0) || (y == height - 1) || (x == width - 1))
		dst[y*width + x] = { 0 };
	else {
		for (int x1 = 0; x1 < WINDOW_SIZE; x1++) {
			for (int y1 = 0; y1 < WINDOW_SIZE; y1++){
				filter[x1*WINDOW_SIZE + y1] = src[(y + y1 - 1)*width + x + x1 - 1];   // setup the filterign window.
			}
		}
		uchar3 temp = sort(filter, 9);
		setValue(dst, width, x, y, temp);
	}
}
void Median_Filter_caller(const uchar* src, uchar* dst, int width, int height)
{
	dim3 threads(16, 16, 1);
	dim3 grids((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y, 1);
	Median_Filter_kernel << <grids, threads >> >(src, dst, width, height);
	//hipDeviceSynchronize();
}

//int main()
//{
//
//	//test1();
//	for (int i = 0; i < 4; i++)
//		test2();
//
//	return 0;
//}

void test1(cv::Mat& img)
{
	using namespace cv;
	typedef Mat mat;
	mat dst;
	size_t memSize = img.step * img.rows;
	uchar* d_src = nullptr;
	uchar* d_dst = nullptr;
	hipMalloc((void**)&d_src, memSize);
	hipMalloc((void**)&d_dst, memSize);
	hipMemcpy(d_src, img.data, memSize, hipMemcpyHostToDevice);
	Median_Filter_caller(d_src, d_dst, img.cols, img.rows);
	hipMemcpy(img.data, d_dst, memSize, hipMemcpyDeviceToHost);
}
void test2()
{

	cv::Mat image = cv::imread("1.bmp");
	cv::Mat dst;
	/*cv::imshow("src", image);*/
	salt(image, 3000);
	pepper(image, 3000);
	/*cv::imshow("noise", image);*/
	MyTimer t1;
	t1.start();
	cv::medianBlur(image, dst, 3);
	t1.stop();
	std::cout << "CPU Time  is: " << t1.elapse() << " ms" << std::endl << std::endl;
	/*cv::imshow("CPU",dst);*/
	size_t memSize = image.step * image.rows;
	uchar* d_src = nullptr;
	uchar* d_dst = nullptr;

	hipMalloc((void**)&d_src, memSize);
	hipMalloc((void**)&d_dst, memSize);
	MyTimer t2;

	hipMemcpy(d_src, image.data, memSize, hipMemcpyHostToDevice);
	t2.start();
	Median_Filter_caller(d_src, d_dst, image.cols, image.rows);

	hipMemcpy(image.data, d_dst, memSize, hipMemcpyDeviceToHost);
	t2.stop();
	std::cout << " GPU Time  is: " << t2.elapse() << " ms" << std::endl << std::endl;
	/*cv::imshow("gpu", image);
	cv::waitKey(0);*/
	hipFree(d_src);
	hipFree(d_dst);
}

